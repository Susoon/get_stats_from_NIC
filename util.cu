#include "hip/hip_runtime.h"
#include "util.cu.h"
#include "eth.cu.h"
#include "arp.cu.h" 
#include "icmp.cu.h"
#include "ip.cu.h"
#include "log.h"
#include <linux/ip.h>
#include <linux/udp.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>

//#include <sys/types.h>
//#include <sys/socket.h>
#include <arpa/inet.h> // IPPROTO_TCP, IPPROTO_ICMP

// returns a timestamp in nanoseconds
// based on rdtsc on reasonably configured systems and is hence fast
uint64_t monotonic_time() {
	struct timespec timespec;
	clock_gettime(CLOCK_MONOTONIC, &timespec);
	return timespec.tv_sec * 1000 * 1000 * 1000 + timespec.tv_nsec;
}

uint32_t get_pkt_size_idx(uint32_t pkt_size)
{
    uint32_t idx = 0;

    switch(pkt_size){
        case 64:
            idx = 0;
            break;
        case 127:
            idx = 1;
            break;
        case 255:
            idx = 2;
            break;
        case 511:
            idx = 3;
            break;
        case 1023:
            idx = 4;
            break;
        case 1522:
            idx = 5;
            break;
        case 4096:
            idx = 6;
            break;
        default:
            idx = 0;
            break;
    }

    return idx;
}

void update_memcpy(uint32_t** pkt_cnt, uint32_t** pkt_size, uint32_t *prev_pkt, uint32_t *cur_pkt)
{
    char units[] = {' ', 'K', 'M', 'G', 'T'};
    double pkts[2];
    char pps[2][40];
    char bps[2][40];
    uint32_t p_size = 0;
    int i, j;
#if 1
    ASSERTRT(hipMemcpy(&cur_pkt[0], &(*pkt_cnt)[0], sizeof(uint32_t), hipMemcpyDeviceToHost));
    ASSERTRT(hipMemcpy(&cur_pkt[1], &(*pkt_cnt)[1], sizeof(uint32_t), hipMemcpyDeviceToHost));
#else

    ASSERTRT(hipMemcpy(&cur_pkt[0], pkt_cnt[0], sizeof(int), hipMemcpyDeviceToHost));
    ASSERTRT(hipMemcpy(&cur_pkt[1], pkt_cnt[1], sizeof(int), hipMemcpyDeviceToHost));
#endif
    ASSERTRT(hipMemcpy(&p_size, *pkt_size, sizeof(uint32_t), hipMemcpyDeviceToHost));
    p_size += 4;

    system("clear");	
    printf("[Memcpy] Using hipMemcpy\n");
#if 0
    printf("[CKJUNG] buf #0\n");
    for(i = 0; i < 1024; i++){
        printf("%d ", data[i]);
    }
    printf("\n\n");
#endif
    for(i = 0; i < 2; i++){
        double tmp_pps;
        double tmp;
        //double batch;
        if (prev_pkt[i] != cur_pkt[i]){ // If we got a traffic flow
            //printf("prev != cur________________prev_pkt[%d]: %d, cur_pkt[%d]: %d\n", i, prev_pkt[i], i, cur_pkt[i]);
            pkts[i] = (double)(cur_pkt[i] - prev_pkt[i]);

#if 0
            if(i == 0)
                printf("RX_pkts: %d\n", (int)pkts[i]); 
            else
                printf("TX_pkts: %d\n", (int)pkts[i]); 
#endif
            tmp = tmp_pps = pkts[i];
            //batch = tmp/BATCH;
            for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
                tmp /= 1000;
            sprintf(pps[i],"%.3lf %c" ,tmp, units[j]);
#if 0
            p_size = PKT_SIZE;
#endif

            //tmp = pkts[i] * p_size * 8; // Bytes -> Bits
            tmp = pkts[i] * p_size * 8 + tmp_pps * 20 * 8; // Add IFG also, 20.01.15, CKJUNG
            for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
                tmp /= 1000;

            double percent = 100.0;
            percent = tmp/percent*100;
            sprintf(bps[i],"%.3lf %c" ,tmp, units[j]);

            if(i == 0){
                //printf("[RX] pps: %spps %sbps(%.2lf %), pkt_size: %d \n", pps[i], bps[i], percent, p_size);
                printf("[RX] pps: %spps %sbps(", pps[i], bps[i]);
                if(percent >= 99){
                    START_GRN
                        printf("%.2lf %%",percent);
                    END
                }else{
                    START_YLW
                        printf("%.2lf %%",percent);
                    END
                }
                printf("), pkt_size: ");
                START_RED
                    printf("%d \n", p_size);
                END
            }else{
                /*
                   printf("[TX] pps: %spps %sbps(%.2lf %%), pkt_size: ", pps[i], bps[i], percent);
                 */

                printf("[TX] pps: %spps %sbps(", pps[i], bps[i]);
                if(percent >= 99){
                    START_GRN
                        printf("%.2lf %%",percent);
                    END
                }else{
                    START_YLW
                        printf("%.2lf %%",percent);
                    END
                }
                printf("), pkt_size: ");
                START_RED
                    printf("%d \n", p_size);
                END
            }
        }else{
            if(i == 0)
                printf("[RX] pps: None\n");
            else
                printf("[TX] pps: None\n");
        }
    }
#if 0
    for(i = 0; i<STATUS_SIZE; i++)
    {
        if(i % 512 ==0)
            printf("\n\n");
        if(buf_idx[i] == 1){
            START_GRN
                printf("%d ", buf_idx[i]);
            END
        }else if(buf_idx[i] == 2){
            START_RED
                printf("%d ", buf_idx[i]);
            END
        }else if(buf_idx[i] == 3){
            START_BLU
                printf("%d ", buf_idx[i]);
            END
        }else{
            printf("%d ", buf_idx[i]);
        }
    }
    printf("\n");
#endif
    prev_pkt[0] = cur_pkt[0];
    prev_pkt[1] = cur_pkt[1];

    printf("\n");
}


void update_stats(
        uint8_t *bar_addr, 
        uint32_t port, 
        struct ice_stats *prev_stats, 
        struct ice_stats *cur_stats)
{
    char units[] = {' ', 'K', 'M', 'G', 'T'};
    char *pkt_sizes[] = {"64", "65-127", "128-255", "256-511", "512-1023", "1024-1522", "big"};
    double pkts;
    char pps[40];
    char bps[40];
    uint32_t pkt_size[2] = {0};
    int i;

    ice_stat_update_rx(bar_addr, port, &prev_stats->rx_stats, &cur_stats->rx_stats, &pkt_size[0]);
    ice_stat_update_tx(bar_addr, port, &prev_stats->tx_stats, &cur_stats->tx_stats, &pkt_size[1]);
    
/*
    printf("[%s] rx : %ld tx :%ld\n", __FUNCTION__, cur_stats->rx_stats.rx_total, cur_stats->tx_stats.tx_total);
    printf("[%s] rx : %ld tx :%ld\n", __FUNCTION__, prev_stats->rx_stats.rx_total, prev_stats->tx_stats.tx_total);
*/

    //system("clear");	

    double tmp_pps;
    double tmp;

    printf("[STATS] Using NIC stats\n");

    // If we got a RX traffic flow
    if (prev_stats->rx_stats.rx_total != cur_stats->rx_stats.rx_total){ 
        pkts = (double)(cur_stats->rx_stats.rx_total - prev_stats->rx_stats.rx_total);

        tmp = tmp_pps = pkts;
        for(i = 0; tmp >= 1000 && i < sizeof(units)/sizeof(char) -1; i++)
            tmp /= 1000;
        sprintf(pps, "%.3lf %c" ,tmp, units[i]);

        tmp = pkts * pkt_size[0] * 8 + tmp_pps * 20 * 8; // Add IFG also, 20.01.15, CKJUNG
        tmp = (cur_stats->rx_stats.rx_bytes - prev_stats->rx_stats.rx_bytes) * 8 + tmp_pps * 20 * 8;
        for(i = 0; tmp >= 1000 && i < sizeof(units)/sizeof(char) -1; i++)
            tmp /= 1000;

        double percent = 100.0;
        percent = tmp/percent*100;
        sprintf(bps, "%.3lf %c" ,tmp, units[i]);

        printf("[RX] pps: %spps %sbps(", pps, bps);
        if(percent >= 99){
            START_GRN
                printf("%.2lf %%",percent);
            END
        }else{
            START_YLW
                printf("%.2lf %%",percent);
            END
        }
        printf("), pkt_size: ");
        START_RED
            printf("%s \n", pkt_sizes[get_pkt_size_idx(pkt_size[0])]);
        END
    }
    else{
        printf("[RX] pps: None\n");
    }

    if(prev_stats->tx_stats.tx_total != cur_stats->tx_stats.tx_total){ 
        pkts = (double)(cur_stats->tx_stats.tx_total - prev_stats->tx_stats.tx_total);

        tmp = tmp_pps = pkts;
        for(i = 0; tmp >= 1000 && i < sizeof(units)/sizeof(char) -1; i++)
            tmp /= 1000;
        sprintf(pps, "%.3lf %c" ,tmp, units[i]);

        tmp = pkts * pkt_size[1] * 8 + tmp_pps * 20 * 8; // Add IFG also, 20.01.15, CKJUNG
        tmp = (cur_stats->tx_stats.tx_bytes - prev_stats->tx_stats.tx_bytes) * 8 + tmp_pps * 20 * 8;
        for(i = 0; tmp >= 1000 && i < sizeof(units)/sizeof(char) -1; i++)
            tmp /= 1000;

        double percent = 100.0;
        percent = tmp/percent*100;
        sprintf(bps, "%.3lf %c" ,tmp, units[i]);

        printf("[TX] pps: %spps %sbps(", pps, bps);
        if(percent >= 99){
            START_GRN
                printf("%.2lf %%", percent);
            END
        }else{
            START_YLW
                printf("%.2lf %%", percent);
            END
        }
        printf("), pkt_size: ");
        START_RED
            printf("%s \n", pkt_sizes[get_pkt_size_idx(pkt_size[1])]);
        END
    }
    else{
        printf("[TX] pps: None\n");
    }

    printf("\n");
}

void monitoring_loop(uint8_t *bar_addr, uint32_t** pkt_cnt, uint32_t** pkt_size)
{
    START_GRN
        printf("[Monitoring] Control is returned to CPU!\n");
    END

        struct ice_stats *cur_stats;
    struct ice_stats *prev_stats;

    uint32_t prev_pkt[2] = {0,}, cur_pkt[2] = {0,};
    int elapsed_time = 0;

    uint64_t last_stats_printed = monotonic_time();
    uint64_t time;

    cur_stats = (struct ice_stats *)calloc(1, sizeof(struct ice_stats));
    prev_stats = (struct ice_stats *)calloc(1, sizeof(struct ice_stats));

    while(1)                                           
    {
        time = monotonic_time();
        if(time - last_stats_printed > 1000 * 1000 * 1000){
            elapsed_time++; // 1 sec +
            last_stats_printed = time;

            update_memcpy(pkt_cnt, pkt_size, prev_pkt, cur_pkt);
            update_stats(bar_addr, 0, prev_stats, cur_stats);

            int second = elapsed_time%60;
            int minute = elapsed_time%3600/60;
            int hour   = elapsed_time/3600;

            printf("Elapsed: %3d h %3d m %3d s\n(ctrl + c) to stop.\n", hour, minute, second);
        }
        //sleep(1); 
    }                                                                  
}




__device__ void DumpPacket_raw(unsigned char* buf, int len)
{
    int i;

	START_YLW
	printf("[START]___________________________________________\n");
	END
	printf("DumpPkt_____________________________________HEX___\n");
	for(i = 0; i < len; i++)
	{
		if(i % 16 == 0)
			printf("\n");

		printf("%02x ", buf[i]);
	}
	printf("\n____________________________________________HEX___\n\n");

	START_YLW
	printf("[END]___________________________________________\n\n\n");
	END
}

__device__ void DumpARPPacket(struct arphdr *arph)
//void DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}

__device__ void 
DumpICMPPacket(const char* type, struct icmphdr *icmph, uint32_t saddr, uint32_t daddr)
{
  uint8_t* _saddr = (uint8_t*) &saddr;
  uint8_t* _daddr = (uint8_t*) &daddr;

	printf("ICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));

  printf("Sender IP: %u.%u.%u.%u\n",
      *_saddr++, *_saddr++, *_saddr++, *_saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *_daddr++, *_daddr++, *_daddr++, *_daddr);

  printf("%s--------------------------------------------\n", type);
  for(int i=0; i<64; i+=2) {
    printf("%x ", *(((uint8_t*)icmph) + i));
    printf("%x ", *(((uint8_t*)icmph) + i+1));
    if(i%20==0)
      printf("\n");
  }
  printf("\n--------------------------------------------\n");
}

__device__ void 
DumpICMPPacket(struct icmphdr *icmph, uint32_t saddr, uint32_t daddr)
{
  uint8_t* _saddr = (uint8_t*) &saddr;
  uint8_t* _daddr = (uint8_t*) &daddr;

	printf("ICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));

  printf("Sender IP: %u.%u.%u.%u\n",
      *_saddr++, *_saddr++, *_saddr++, *_saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *_daddr++, *_daddr++, *_daddr++, *_daddr);

  printf("--------------------------------------------\n");
  for(int i=0; i<100; i+=2) {
    printf("%x ", *(((uint8_t*)icmph) + i));
    printf("%x ", *(((uint8_t*)icmph) + i+1));
    if(i%20==0)
      printf("\n");
  }
  printf("\n--------------------------------------------\n");
}

__device__ void 
DumpICMPPacket(struct icmphdr *icmph, uint8_t* saddr, uint8_t* daddr)
{
	printf("\nICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));
	printf("ICMP_checksum: 0x%x\n", icmph->icmp_checksum);
  printf("Sender IP: %u.%u.%u.%u\n",
      *saddr++, *saddr++, *saddr++, *saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *daddr++, *daddr++, *daddr++, *daddr);
}

__device__ void DumpPacket(uint8_t *buf, int len)
{
  printf("\n\n\n<<<DumpPacket>>>----------------------------------------\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      DumpARPPacket((struct arphdr *) (ethh + 1));
	//	goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));
	else if (iph->protocol == IPPROTO_ICMP){
		struct icmphdr *icmph = (struct icmphdr *) IP_NEXT_PTR(iph);
		DumpICMPPacket(icmph, (uint8_t*)&(iph->saddr), (uint8_t*)&(iph->daddr));
	}

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
  printf("<<<DumpPacket>>>-----------------------------------END--\n");

}

__device__ void schedule_tester(unsigned int sch[], unsigned int *count) {
	if(threadIdx.x == 0){
		if((*count)++ == 10000000){
		//if((*count)++ == 100000){
			printf("--------------------------------------------\n");
			for(int i = 0; i < 512; i++){                            
				if(i % 32 == 0)                                        
					printf("\n");                                        
				printf("tid: %4d, cnt: %d\n", i, sch[i]);              
			}                                                        
			printf("------------------------------------------\n\n");
			*count = 0;
		}
	}

}
